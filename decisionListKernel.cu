
#include <hip/hip_runtime.h>
typedef enum {DT_UNDEFINED = 0, DT_INT, DT_FLOAT, DT_STRING, DT_BOOLEAN} DataType;

typedef enum {OP_UNDEFINED = 0, OP_EQUAL_TO, OP_GREATER_THAN, OP_GREATER_THAN_OR_EQUAL_TO, OP_LESS_THAN, 
    OP_LESS_THAN_OR_EQUAL_TO, OP_LOGICAL_AND, OP_LOGICAL_OR, OP_NOT_EQUAL_TO} Operator;

typedef enum {DL_ERROR = 0, DL_FALSE = 1, DL_TRUE = 2, DL_IGNORE = 3} DLNodeValue;

typedef struct {
    DataType type;
    int intValue;
    float floatValue;
    char *stringValue;
    int booleanValue;
} Value;

#define OFFSET_SAFETY_MAX 100

__device__ int parseDecisionListNode(char *expression, DLNodeValue *value);
__device__ int parseExpression(char *expression, Value *value);
__device__ int parseBinaryExpression(char *expression, Value *value);
__device__ int parseVariableExpression(char *expression, Value *value);
__device__ int parseBooleanConstant(char *expression, Value *value);
__device__ int parseIntegerConstant(char *expression, Value *value);
__device__ int parseFloatConstant(char *expression, Value *value);
__device__ int parseStringConstant(char *expression, Value *value);
__device__ int parseOperator(char *expression, Operator *op);
__device__ void evaluateBinaryExpression(Value *operand1, Operator op, Value *operand2, Value *returnValue);
__device__ void evaluateIntegerComparison(int op1, Operator op, int op2, Value *value);
__device__ void evaluateFloatComparison(float op1, Operator op, float op2, Value *value);
__device__ void evaluateStringComparison(char *op1, Operator op, char *op2, Value *value);
__device__ void evaluateBooleanComparison(int op1, Operator op, int op2, Value *value);
__device__ int dstrlen(char *str);
__device__ int dstreql(char *str1, char *str2);

extern "C"
__global__ void processDecisionLists(int numExpressions, char **expressions, int *output)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numExpressions) {
        char *expression = expressions[idx];
        DLNodeValue dlNodeValue;
        int offset = parseDecisionListNode(expression, &dlNodeValue);
        output[idx] = dlNodeValue;
    }
}

/////////////////////////////////////////////////////////////////////
// PARSING FUNCTIONS
//
// NB: All the parseXXX functions return a value that indicates how far the pointer 
// should be advanced.  The actual return value is in the parameter list.

__device__ int parseDecisionListNode(char *expression, DLNodeValue *dlNodeValue) {
    
    // Currently there are only two valid formats for a DL node:
    //      <binary expression> <T|F>
    //      <boolean constant> <T|F>
    // In the latter case, the <boolean constant> must always be T since that represents
    // the default node.  It's redundant to have a condition that always evaluates to true,
    // but we keep it anyway because the code to generate, store and evaluate DL's on the 
    // Java side is much nicer that way.

    int offset = 0;
    Value value;
    
    offset += parseExpression(expression, &value);

    // Check the return from the expression evaluation.  If it's false, then we ignore this
    // DL node and move on to the next one (so return IGNORE); if true, then we return the
    // node's value.
    if (value.type != DT_BOOLEAN) {
        *dlNodeValue = DL_ERROR;
        return 0;
    }
    if (value.booleanValue == 0) {
        *dlNodeValue = DL_IGNORE; 
     } else {
        char nodeValue = *(expression+offset);
        if (nodeValue == 'T')
            *dlNodeValue = DL_TRUE;
        else if (nodeValue == 'F')
            *dlNodeValue = DL_FALSE;
        else {
            *dlNodeValue = DL_ERROR;
            return 0;
        }
     } 
    
    return offset;
}

__device__ int parseExpression(char *expression, Value *value) {
    int offset = 0;

    char c1 = expression[0];
    char c2 = expression[1];
    offset += 2;
    
    // NB: This is where you'd plug in the code to evaluate additional kinds of expressions
    // if you wanted to expand this kernel to be more generic.
    
    if (c1 == 'E' && c2 == 'B')
        offset += parseBinaryExpression(expression+offset, value);
    else if (c1 == 'E' && c2 == 'V')
        offset += parseVariableExpression(expression+offset, value);
    else if (c1 == 'C' && c2 == 'B')
        offset += parseBooleanConstant(expression+offset, value);
    else if (c1 == 'C' && c2 == 'I')
        offset += parseIntegerConstant(expression+offset, value);
    else if (c1 == 'C' && c2 == 'F')
        offset += parseFloatConstant(expression+offset, value);
    else if (c1 == 'C' && c2 == 'S')
        offset += parseStringConstant(expression+offset, value);
    else { // error
        value->type = DT_UNDEFINED;
        return 0;
    }

    return offset;
}

__device__ int parseBinaryExpression(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (*expression != '{')
        return 0;
    offset++;

    Value operand1;
    Operator op;
    Value operand2;
    offset += parseExpression(expression+offset, &operand1);
    offset += parseOperator(expression+offset, &op);
    offset += parseExpression(expression+offset, &operand2);

    // Evaluate the binary expression
    evaluateBinaryExpression(&operand1, op, &operand2, value);

    // Skip over closing }    
    if (*(expression+offset) != '}') {
        value->type = DT_UNDEFINED;
        return 0;
    }    
    offset++;

    return offset;
}

__device__ int parseVariableExpression(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (*expression != '{')
        return 0;
    offset++;

    char *token = expression+offset;
    while (*(expression+offset) != '}' && offset < OFFSET_SAFETY_MAX)
        offset++;
    if (offset == OFFSET_SAFETY_MAX)
        return 0;
    *(expression+offset) = '\0';
    offset++;

   
    // TODO: Look up variable in symbol table.
    // Of course, to do that we need to *have* a symbol table, so that's first on the list.


    return offset;
}

__device__ int parseBooleanConstant(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (*expression != '{')
        return 0;
    offset++;

    if (*(expression+offset) == 'F') {
        value->booleanValue = 0;
        value->type = DT_BOOLEAN;
    } else if (*(expression+offset) == 'T') {
        value->booleanValue = 1;
        value->type = DT_BOOLEAN;
    } else { // error
        value->type = DT_UNDEFINED;
        return 0; 
    }
    offset++;
    
    // Skip over closing }    
    if (*(expression+offset) != '}')
        return 0;
    offset++;

    return offset;
}

__device__ int parseIntegerConstant(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (*expression != '{')
        return 0;
    offset++;

    value->intValue = 0;
    while (*(expression+offset) != '}' && offset < OFFSET_SAFETY_MAX) {
        value->intValue = value->intValue * 10 + (*(expression+offset) - '0');  
        offset++;
    }
    if (offset == OFFSET_SAFETY_MAX)
        return 0;
    value->type = DT_INT;
    offset++;

    return offset;
}

__device__ int parseFloatConstant(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (expression[0] != '{')
        return 0;
    offset++;

    if (*(expression+offset) != '0')
        return 0;
    offset++;
    if (*(expression+offset) != '.')
        return 0;
    offset++;
    value->floatValue = 0;
    int divisor = 10;
    while (*(expression+offset) != '}' && offset < OFFSET_SAFETY_MAX) {
        value->floatValue = value->floatValue + ((float)(*(expression+offset) - '0'))/divisor; 
        divisor = divisor * 10;
        offset++;
    }
    if (offset == OFFSET_SAFETY_MAX)
        return 0;
    value->type = DT_FLOAT;
    offset++;

    return offset;
}

__device__ int parseStringConstant(char *expression, Value *value) {
    int offset = 0;

    // Skip over opening {
    if (*expression != '{')
        return 0;
    offset++;

    char *token = expression+offset;
    while (*(expression+offset) != '}' && offset < OFFSET_SAFETY_MAX)
        offset++;
    if (offset == OFFSET_SAFETY_MAX)
        return 0;
    *(expression+offset) = '\0';
    offset++;
    
    value->type = DT_STRING;
    value->stringValue = token; 

    return offset;
}

__device__ int parseOperator(char *expression, Operator *op) {
    char c1 = expression[0];
    char c2 = expression[1];
    
    if (c1 == '=' && c2 == '=')
        *op = OP_EQUAL_TO;
    else if (c1 == '>' && c2 == '>')
        *op = OP_GREATER_THAN;
    else if (c1 == '>' && c2 == '=')
        *op = OP_GREATER_THAN_OR_EQUAL_TO;
    else if (c1 == '<' && c2 == '<')
        *op = OP_LESS_THAN;
    else if (c1 == '<' && c2 == '=')
        *op = OP_LESS_THAN_OR_EQUAL_TO;
    else if (c1 == '&' && c2 == '&')
        *op = OP_LOGICAL_AND;
    else if (c1 == '|' && c2 == '|')
        *op = OP_LOGICAL_OR;
    else if (c1 == '!' && c2 == '=')
        *op = OP_NOT_EQUAL_TO;
    else // error
        return 0;
    
    return 2;
}

/////////////////////////////////////////////////////////////////////
// EVALUATION FUNCTIONS

__device__ void evaluateBinaryExpression(Value *operand1, Operator op, Value *operand2, Value *value) {
    // Indicate an error by not setting the type on the return value
    value->type = DT_UNDEFINED;
    
    // For now only allowing comparison of the same types
    if (operand1->type != operand2->type)
        return;
    switch (operand1->type) {
        case DT_INT:
            evaluateIntegerComparison(operand1->intValue, op, operand2->intValue, value);
            break;
        case DT_FLOAT:
            evaluateFloatComparison(operand1->floatValue, op, operand2->floatValue, value);
            break;
        case DT_STRING:
            evaluateStringComparison(operand1->stringValue, op, operand2->stringValue, value);
            break;
        case DT_BOOLEAN:
            evaluateBooleanComparison(operand1->booleanValue, op, operand2->booleanValue, value);
            break;
        default:
        case DT_UNDEFINED:
            // do nothing
            break;
    }
}

__device__ void evaluateIntegerComparison(int op1, Operator op, int op2, Value *value) {
    value->type = DT_BOOLEAN;
    int bv = 0;  // assume comparison is false
    switch (op) {
        case OP_EQUAL_TO:
            if (op1 == op2) bv = 1;
            break;
        case OP_GREATER_THAN:
            if (op1 > op2) bv = 1;
            break;
        case OP_GREATER_THAN_OR_EQUAL_TO:
            if (op1 >= op2) bv = 1;
            break;
        case OP_LESS_THAN:
            if (op1 < op2) bv = 1;
            break;
        case OP_LESS_THAN_OR_EQUAL_TO:
            if (op1 <= op2) bv = 1;
            break;
        case OP_LOGICAL_AND:
            bv = op1 && op2;
            break;
        case OP_LOGICAL_OR:
            bv = op1 || op2;
            break;
        case OP_NOT_EQUAL_TO:
            if (op1 != op2) bv = 1;
            break;
        default:
        case OP_UNDEFINED:
            break;
    }
    value->booleanValue = bv;
}

__device__ void evaluateFloatComparison(float op1, Operator op, float op2, Value *value) {
    value->type = DT_BOOLEAN;
    int bv = 0;  // assume comparison is false
    switch (op) {
        case OP_EQUAL_TO:
            if (op1 == op2) bv = 1;
            break;
        case OP_GREATER_THAN:
            if (op1 > op2) bv = 1;
            break;
        case OP_GREATER_THAN_OR_EQUAL_TO:
            if (op1 >= op2) bv = 1;
            break;
        case OP_LESS_THAN:
            if (op1 < op2) bv = 1;
            break;
        case OP_LESS_THAN_OR_EQUAL_TO:
            if (op1 <= op2) bv = 1;
            break;
        case OP_LOGICAL_AND:
            bv = op1 && op2;
            break;
        case OP_LOGICAL_OR:
            bv = op1 || op2;
            break;
        case OP_NOT_EQUAL_TO:
            if (op1 != op2) bv = 1;
            break;
        default:
        case OP_UNDEFINED:
            return;
    }
    value->booleanValue = bv;
}

__device__ void evaluateStringComparison(char *op1, Operator op, char *op2, Value *value) {
    // Because time is short, we'll have to skimp on the string comparisons
    // The greater than and less than operations require a lexical comparison,
    // and we don't have access to the standard C library (and thus strcmp()).
    // I'm not not going to write my own strcmp() function, so equality is the
    // only operation we're going to support for now.
      
    value->type = DT_BOOLEAN;
    int bv = 0;
    switch (op) {
        case OP_EQUAL_TO:
            if (dstreql(op1, op2) == 1) bv = 1;
            break;
        case OP_NOT_EQUAL_TO:
            if (dstreql(op1, op2) == 0) bv = 1;
            break;
        default:
        case OP_GREATER_THAN:
        case OP_GREATER_THAN_OR_EQUAL_TO:
        case OP_LESS_THAN:
        case OP_LESS_THAN_OR_EQUAL_TO:
        case OP_LOGICAL_AND:
        case OP_LOGICAL_OR:
        case OP_UNDEFINED:
            break;
    }
    value->booleanValue = bv;
}

__device__ void evaluateBooleanComparison(int op1, Operator op, int op2, Value *value) {
    value->type = DT_BOOLEAN;
    int bv = 0;
    switch (op) {
        case OP_EQUAL_TO:
            if (op1 == op2) bv = 1;
            break;
        case OP_LOGICAL_AND:
            bv = op1 && op2;
            break;
        case OP_LOGICAL_OR:
            bv = op1 || op2;
            break;
        case OP_NOT_EQUAL_TO:
            if (op1 != op2) bv = 1;
            break;
        default:
        case OP_GREATER_THAN:
        case OP_GREATER_THAN_OR_EQUAL_TO:
        case OP_LESS_THAN:
        case OP_LESS_THAN_OR_EQUAL_TO:
        case OP_UNDEFINED:
            break;
    }
    value->booleanValue = bv;
}

/////////////////////////////////////////////////////////////////////
// STRING FUNCTIONS

__device__ int dstrlen(char *str) {
    int len = 0;
    while (*str != '\0') {
        str++;
        len++;
    }
    return len;        
}

__device__ int dstreql(char *str1, char *str2) {
    while (*str1 == *str2 && *str1 != '\0' && *str2 != '\0') {
        str1++;
        str2++;
    }
    if (*str1 == '\0' && *str2 == '\0')
        return 1;
    return 0;
}
